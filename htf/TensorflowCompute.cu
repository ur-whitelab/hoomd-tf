#include "hip/hip_runtime.h"
// Copyright (c) 2018 Andrew White at the University of Rochester
//  This file is part of the Hoomd-Tensorflow plugin developed by Andrew White

#include "TensorflowCompute.cuh"
#include <iostream>


/*! \file TensorflowCompute.cu
    \brief CUDA kernels and functions for TensorflowCompute
*/

extern "C" __global__
void gpu_add_scalar4_kernel(Scalar4 *dest, Scalar4 *src, unsigned int N)
    {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
        {
        dest[i].x += src[i].x;
        dest[i].y += src[i].y;
        dest[i].z += src[i].z;
        dest[i].w += src[i].w;
        }
    }

hipError_t gpu_add_scalar4(Scalar4 *dest, Scalar4 *src, unsigned int _N, hipStream_t s)
    {
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)_N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_add_scalar4_kernel<<< grid, threads, 0, s >>>(dest, src, _N);

    // this method always succeds.
    // If you had a cuda* call in this driver, you could return its error code, if not
    // hipSuccess
    return hipSuccess;
    }

extern "C" __global__
void gpu_add_virial_kernel(Scalar *dest, Scalar *src, unsigned int _N, unsigned int _pitch)
    {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < _N)
        {
        dest[0 * _pitch + i] += src[i * 9 + 0]; //xx
        dest[1 * _pitch + i] += src[i * 9 + 1]; //xy
        dest[2 * _pitch + i] += src[i * 9 + 2]; //xz
        dest[3 * _pitch + i] += src[i * 9 + 4]; //yy
        dest[4 * _pitch + i] += src[i * 9 + 5]; //yz
        dest[5 * _pitch + i] += src[i * 9 + 8]; //zz
        }
    }

hipError_t gpu_add_virial(Scalar *dest, Scalar *src, unsigned int _N, unsigned int _pitch, hipStream_t s)
    {
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)_N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_add_virial_kernel<<< grid, threads, 0, s >>>(dest, src, _N, _pitch);

    // this method always succeds.
    // If you had a cuda* call in this driver, you could return its error code, if not
    // hipSuccess
    return hipSuccess;
    }

#include "hoomd/TextureTools.h"
#include "hoomd/Index1D.h"
#include <assert.h>

//! Texture for reading particle positions
scalar4_tex_t pdata_pos_tex;

//! Texture for reading the neighbor list
texture<unsigned int, 1, hipReadModeElementType> nlist_tex;

template<unsigned char use_gmem_nlist>
__global__ void gpu_reshape_nlist_kernel(Scalar4* dest,
                                         const unsigned int N,
                                         const unsigned int NN,
                                         const unsigned int offset,
                                         const unsigned int batch_size,
                                         const Scalar4 *d_pos,
                                         const BoxDim box,
                                         const unsigned int *d_n_neigh,
                                         const unsigned int *d_nlist,
                                         const unsigned int *d_head_list,
                                         double rmax)
    {

    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (idx >= N || idx - offset >= batch_size)
        return;

    // load in the length of the list
    unsigned int n_neigh = d_n_neigh[idx];
    const unsigned int head_idx = d_head_list[idx];

    // read in the position of our particle. Texture reads of Scalar4's are faster than global reads on compute 1.0 hardware
    Scalar4 postype = texFetchScalar4(d_pos, pdata_pos_tex, idx);
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int typei = __scalar_as_int(postype.w);

    // prefetch neighbor index
    unsigned int cur_neigh = 0;
    unsigned int next_neigh(0);
    if (use_gmem_nlist)
        next_neigh = d_nlist[head_idx];
    else
        next_neigh = texFetchUint(d_nlist, nlist_tex, head_idx);

    // loop over neighbors
    assert(n_neigh <= NN);
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {

        // read the current neighbor index
        // prefetch the next value and set the current one
        cur_neigh = next_neigh;
        if (use_gmem_nlist)
            next_neigh = d_nlist[head_idx + neigh_idx + 1];
        else
            next_neigh = texFetchUint(d_nlist, nlist_tex, head_idx + neigh_idx+1);

        // get the neighbor's position
        Scalar4 neigh_postype = texFetchScalar4(d_pos, pdata_pos_tex, cur_neigh);
        Scalar3 neigh_pos = make_scalar3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

        // calculate dr (with periodic boundary conditions)
        Scalar3 dx = neigh_pos - pos;

        // apply periodic boundary conditions
        dx = box.minImage(dx);

        // access needed parameters
        unsigned int typej = __scalar_as_int(neigh_postype.w);

        // calculate r
        Scalar rsq = dot(dx, dx);

        if (rsq < (rmax * rmax))
        {
            dest[(idx - offset) * NN + neigh_idx].x = dx.x;
            dest[(idx - offset) * NN + neigh_idx].y = dx.y;
            dest[(idx - offset) * NN + neigh_idx].z = dx.z;
            dest[(idx - offset) * NN + neigh_idx].w = static_cast<Scalar> (typej);

            }
        }
    }


hipError_t gpu_reshape_nlist(Scalar4* dest,
			      const Scalar4 *d_pos,
			      const unsigned int N,
                  	      const unsigned int NN,
                  	      const unsigned int offset,
                  	      const unsigned int batch_size,
			      const unsigned int n_ghost,
			      const BoxDim& box,
			      const unsigned int *d_n_neigh,
			      const unsigned int *d_nlist,
			      const unsigned int *d_head_list,
			      const unsigned int size_nlist,
			      const unsigned int block_size,
			      const unsigned int compute_capability,
			      const unsigned int max_tex1d_width,
			      double rmax,
			      hipStream_t stream)
    {

    assert(d_pos);
    assert(dest);
    assert(d_n_neigh);
    assert(d_nlist);
    assert(d_head_list);

    //set neighbors to zeros
    hipMemset(dest, 1, batch_size * NN * sizeof(Scalar4));

    // texture bind
    if (compute_capability < 350)
        {
        // bind the pdata position texture
        pdata_pos_tex.normalized = false;
        pdata_pos_tex.filterMode = hipFilterModePoint;
        hipError_t error = hipBindTexture(0,
                                            pdata_pos_tex,
                                            d_pos,
                                            sizeof(Scalar4) * (N+n_ghost));
        if (error != hipSuccess)
            return error;

        if (size_nlist <= max_tex1d_width)
            {
            nlist_tex.normalized = false;
            nlist_tex.filterMode = hipFilterModePoint;
            error = hipBindTexture(0, nlist_tex, d_nlist, sizeof(unsigned int)*size_nlist);
            if (error != hipSuccess)
                return error;
            }
        }

    if (compute_capability < 350 && size_nlist > max_tex1d_width)
        {
        // use global memory when the neighbor list must be texture bound,
        // but exceeds the max size of a texture
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_reshape_nlist_kernel<1>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_size);

        // setup the grid to run the kernel
        dim3 grid( batch_size / run_block_size + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        gpu_reshape_nlist_kernel<1><<< grid, threads, 0, stream>>>(dest,
            N,
            NN,
            offset,
            batch_size,
            d_pos,
            box,
            d_n_neigh,
            d_nlist,
            d_head_list,
            rmax);
    }
    else
    {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_reshape_nlist_kernel<0>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_size);

        // setup the grid to run the kernel
        dim3 grid( batch_size / run_block_size + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);
        gpu_reshape_nlist_kernel<0><<< grid, threads, 0, stream>>>(dest,
            N,
            NN,
            offset,
            batch_size,
            d_pos,
            box,
            d_n_neigh,
            d_nlist,
            d_head_list,
            rmax);
    }

    return hipSuccess;

    }
