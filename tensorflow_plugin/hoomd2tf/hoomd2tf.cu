
#include <hip/hip_runtime.h>
// Copyright (c) 2018 Andrew White at the University of Rochester
//  This file is part of the Hoomd-Tensorflow plugin developed by Andrew White

#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU

#include <iostream>
#include "hoomd2tf.h"
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"

using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;

// GPU specialization of actual computation.
template <typename T>
void HOOMD2TFFunctor<GPUDevice, T >::operator()(
    const GPUDevice& d, int size, CommStruct* in_memory,T* out) {

  //cudaEventSynchronize(in_memory.event);
  in_memory->readGPUMemory(out, size * sizeof(T));
}

// Explicitly instantiate functors for the types of OpKernels registered.
template struct HOOMD2TFFunctor<GPUDevice, float> ;
template struct HOOMD2TFFunctor<GPUDevice, double> ;

#endif  // GOOGLE_CUDA
